#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


__global__ void MatrixMulBlockCUDA(double* C, double* A, double* B) {
    int wA = 1024;
    int wB = 1024;
    const int BLOCK_SIZE = 32;
        
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd = aBegin + wA - 1;
    int aStep = BLOCK_SIZE;
    int bBegin = BLOCK_SIZE * bx;
    int bStep = BLOCK_SIZE * wB;
    double Csub = 0;

    for (int a = aBegin, b = bBegin;
        a <= aEnd;
        a += aStep, b += bStep) {
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

__global__ void MatrixMulCUDA(double* C, double* A, double* B) {
    int index = threadIdx.x;
    int oneK = 1024;
    for (int i = 0; i < oneK; i++) {
        for (int j = 0; j < oneK; j++) {
            C[oneK * index + j] += (A[oneK * index + i] * B[oneK * i + j]);
        }
    }
    __syncthreads();
}

void ConstantInit(double* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = i;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(const dim3& dimsA, const dim3& dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(double) * size_A;
    double* h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(double) * size_B;
    double* h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
    hipStream_t stream;

    ConstantInit(h_A, size_A);
    ConstantInit(h_B, size_B);

    // Allocate device memory
    double* d_A, * d_B, * d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsB.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(double);
    double* h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C), mem_size_C));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, stream));


    // Question 5a
    dim3 threadsA(1024);
    dim3 gridA(1);
    MatrixMulCUDA <<< gridA, threadsA, 0, stream >>> (d_C, d_A, d_B);
   
    // Question 5b
    int block_size = 32;
    dim3 threadsB(block_size, block_size);
    dim3 gridB(dimsB.x / threadsB.x, dimsA.y / threadsB.y);
    MatrixMulBlockCUDA <<< gridB, threadsB, 0, stream >>> (d_C, d_A, d_B);

    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    printf("Total time in msec: %f\n", msecTotal);

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    return 0;
}


int main(int argc, char** argv) {

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char**)argv);

    int oneK = 1024;

    dim3 dimsA(oneK, oneK, 1);
    dim3 dimsB(oneK, oneK, 1);

    int matrix_result = MatrixMultiply(dimsA, dimsB);

    exit(matrix_result);
}

